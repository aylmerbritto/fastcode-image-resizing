#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <sstream>
#include <stdio.h>
#include <stdlib.h>
#include <cstdlib>
#include "opencv2/opencv.hpp"

using namespace cv;
using namespace std;

// __constant__ float gtop_left_coeff[16];
// __constant__ float gtop_right_coeff[16];
// __constant__ float gbottom_left_coeff[16];
// __constant__ float gbottom_right_coeff[16];

static int INPUTHEIGHT;
static int INPUTWIDTH;

__global__ void base_kernel(float *input, float *output, int x, int y) {
    int thread_id = threadIdx.x; //get thread id; 0-127
    int block_id = blockIdx.x; // get block id;

    // gtop_left_coeff = {1, 2 / 3.0, 1 / 3.0, 0, 2 / 3.0, 4 / 9.0, 2 / 9.0, 0, 1 / 3.0, 2 / 9.0, 1 / 9.0, 0, 0, 0, 0, 0};
    float gtop_left_coeff[16];
    gtop_left_coeff[0] = 1;
    gtop_left_coeff[1] = 2 / 3.0;
    gtop_left_coeff[2] = 1 / 3.0;
    gtop_left_coeff[3] = 0;
    gtop_left_coeff[4] = 2 / 3.0;
    gtop_left_coeff[5] = 4 / 9.0;
    gtop_left_coeff[6] = 2 / 9.0;
    gtop_left_coeff[7] = 0;
    gtop_left_coeff[8] = 1 / 3.0;
    gtop_left_coeff[9] = 2 / 9.0;
    gtop_left_coeff[10] = 1 / 9.0;
    gtop_left_coeff[11] = 0;
    gtop_left_coeff[12] = 0;
    gtop_left_coeff[13] = 0;
    gtop_left_coeff[14] = 0;
    gtop_left_coeff[15] = 0;
    

    // gtop_right_coeff[16] = {0, 1 / 3.0, 2 / 3.0, 1, 0, 2 / 9.0, 4 / 9.0, 2 / 3.0, 0, 1 / 9.0, 2 / 9.0, 1 / 3.0, 0, 0, 0, 0};
    float gtop_right_coeff[16];
    gtop_right_coeff[0] = 0;
    gtop_right_coeff[1] = 1 / 3.0;
    gtop_right_coeff[2] = 2 / 3.0;
    gtop_right_coeff[3] = 1;
    gtop_right_coeff[4] = 0;
    gtop_right_coeff[5] = 2 / 9.0;
    gtop_right_coeff[6] = 4 / 9.0;
    gtop_right_coeff[7] = 2 / 3.0;
    gtop_right_coeff[8] = 0;
    gtop_right_coeff[9] = 1 / 9.0;
    gtop_right_coeff[10] = 2 / 9.0;
    gtop_right_coeff[11] = 1 / 3.0;
    gtop_right_coeff[12] = 0;
    gtop_right_coeff[13] = 0;
    gtop_right_coeff[14] = 0;
    gtop_right_coeff[15] = 0;
    
    // float gbottom_left_coeff[] = {0, 0, 0, 0, 1 / 3.0, 2 / 9.0, 1 / 9.0, 0, 2 / 3.0, 4 / 9.0, 2 / 9.0, 0, 1, 2 / 3.0, 1 / 3.0, 0};
    float gbottom_left_coeff[16];
    gbottom_left_coeff[0] = 0;
    gbottom_left_coeff[1] = 0;
    gbottom_left_coeff[2] = 0;
    gbottom_left_coeff[3] = 0;
    gbottom_left_coeff[4] = 1/3.0;
    gbottom_left_coeff[5] = 2/9.0;
    gbottom_left_coeff[6] = 1/9.0;
    gbottom_left_coeff[7] = 0;
    gbottom_left_coeff[8] = 2 / 3.0;
    gbottom_left_coeff[9] = 4 / 9.0;
    gbottom_left_coeff[10] = 2 / 9.0;
    gbottom_left_coeff[11] = 0;
    gbottom_left_coeff[12] = 1;
    gbottom_left_coeff[13] = 2 / 3.0;
    gbottom_left_coeff[14] = 1 / 3.0;
    gbottom_left_coeff[15] = 0;

    // float gbottom_right_coeff[] = {0, 0, 0, 0, 0, 1 / 9.0, 2 / 9.0, 1 / 3.0, 0, 2 / 9.0, 4 / 9.0, 2 / 3.0, 0, 1 / 3.0, 2 / 3.0, 1};
    float gbottom_right_coeff[16];
    gbottom_right_coeff[0] = 0;
    gbottom_right_coeff[1] = 0;
    gbottom_right_coeff[2] = 0;
    gbottom_right_coeff[3] = 0;
    gbottom_right_coeff[4] = 0;
    gbottom_right_coeff[5] = 1 / 9.0;
    gbottom_right_coeff[6] = 2 / 9.0;
    gbottom_right_coeff[7] = 1 / 3.0;
    gbottom_right_coeff[8] = 2 / 9.0;
    gbottom_right_coeff[9] = 2 / 9.0;
    gbottom_right_coeff[10] = 4 / 9.0;
    gbottom_right_coeff[11] = 2 / 3.0;
    gbottom_right_coeff[12] = 0;
    gbottom_right_coeff[13] = 1 / 3.0;
    gbottom_right_coeff[14] = 2 / 3.0;
    gbottom_right_coeff[15] = 1;
    
    // 16 x 64 -> 32 x 128
    // int kernel_number = thread_id / 4;
    // 4 input pixel: kernel_number * 2, kernel_number * 2 + 1, kernel_number * 2 + x, kernel_number * 2 + x + 1

    int num_blocks = blockDim.x;
    int num_blocks_per_row = x / 128;

    int block_row = block_id / num_blocks_per_row;
    int block_col = block_id % num_blocks_per_row;
    
    int block_start_idx = (block_row * 32) * (x) + block_col * 128; // input: 4 rows of length x per block
    int block_start_idx_out = (block_row * 64) * (x * 2) + block_col * 256; // output: 8 rows of length 2x per block

    // Base Kernel
    int index = thread_id % 4;
    float top_left_1 = input[block_start_idx + (thread_id / 4) * 2];
    float top_right_1 = input[block_start_idx + (thread_id / 4) * 2 + 1];
    float bottom_left_1 = input[block_start_idx + (thread_id / 4) * 2 + x];
    float bottom_right_1 = input[block_start_idx + (thread_id / 4) * 2 + x + 1];

    float top_left_2 = input[block_start_idx + (thread_id / 4) * 2 + 2 * x];
    float top_right_2 = input[block_start_idx + (thread_id / 4) * 2 + 1 + 2 * x];
    float bottom_left_2 = input[block_start_idx + (thread_id / 4) * 2 + x + 2 * x];
    float bottom_right_2 = input[block_start_idx + (thread_id / 4) * 2 + x + 1 + 2 * x];

    float top_left_3 = input[block_start_idx + (thread_id / 4) * 2 + 4 * x];
    float top_right_3 = input[block_start_idx + (thread_id / 4) * 2 + 1 + 4 * x];
    float bottom_left_3 = input[block_start_idx + (thread_id / 4) * 2 + x + 4 * x];
    float bottom_right_3 = input[block_start_idx + (thread_id / 4) * 2 + x + 1 + 4 * x];

    float top_left_4 = input[block_start_idx + (thread_id / 4) * 2 + 6 * x];
    float top_right_4 = input[block_start_idx + (thread_id / 4) * 2 + 1 + 6 * x];
    float bottom_left_4 = input[block_start_idx + (thread_id / 4) * 2 + x + 6 * x];
    float bottom_right_4 = input[block_start_idx + (thread_id / 4) * 2 + x + 1 + 6 * x];

    float top_left_5 = input[block_start_idx + (thread_id / 4) * 2 + 8 * x];
    float top_right_5 = input[block_start_idx + (thread_id / 4) * 2 + 1 + 8 * x];
    float bottom_left_5 = input[block_start_idx + (thread_id / 4) * 2 + x + 8 * x];
    float bottom_right_5 = input[block_start_idx + (thread_id / 4) * 2 + x + 1 + 8 * x];

    float top_left_6 = input[block_start_idx + (thread_id / 4) * 2 + 10 * x];
    float top_right_6 = input[block_start_idx + (thread_id / 4) * 2 + 1 + 10 * x];
    float bottom_left_6 = input[block_start_idx + (thread_id / 4) * 2 + x + 10 * x];
    float bottom_right_6 = input[block_start_idx + (thread_id / 4) * 2 + x + 1 + 10 * x];

    float top_left_7 = input[block_start_idx + (thread_id / 4) * 2 + 12 * x];
    float top_right_7 = input[block_start_idx + (thread_id / 4) * 2 + 1 + 12 * x];
    float bottom_left_7 = input[block_start_idx + (thread_id / 4) * 2 + x + 12 * x];
    float bottom_right_7 = input[block_start_idx + (thread_id / 4) * 2 + x + 1 + 12 * x];

    float top_left_8 = input[block_start_idx + (thread_id / 4) * 2 + 14 * x];
    float top_right_8 = input[block_start_idx + (thread_id / 4) * 2 + 1 + 14 * x];
    float bottom_left_8 = input[block_start_idx + (thread_id / 4) * 2 + x + 14 * x];
    float bottom_right_8 = input[block_start_idx + (thread_id / 4) * 2 + x + 1 + 14 * x];

    float top_left_9 = input[block_start_idx + (thread_id / 4) * 2 + 16 * x];
    float top_right_9 = input[block_start_idx + (thread_id / 4) * 2 + 1 + 16 * x];
    float bottom_left_9 = input[block_start_idx + (thread_id / 4) * 2 + x + 16 * x];
    float bottom_right_9 = input[block_start_idx + (thread_id / 4) * 2 + x + 1 + 16 * x];

    float top_left_10 = input[block_start_idx + (thread_id / 4) * 2 + 18 * x];
    float top_right_10 = input[block_start_idx + (thread_id / 4) * 2 + 1 + 18 * x];
    float bottom_left_10 = input[block_start_idx + (thread_id / 4) * 2 + x + 18 * x];
    float bottom_right_10 = input[block_start_idx + (thread_id / 4) * 2 + x + 1 + 18 * x];

    float top_left_11 = input[block_start_idx + (thread_id / 4) * 2 + 20 * x];
    float top_right_11 = input[block_start_idx + (thread_id / 4) * 2 + 1 + 20 * x];
    float bottom_left_11 = input[block_start_idx + (thread_id / 4) * 2 + x + 20 * x];
    float bottom_right_11 = input[block_start_idx + (thread_id / 4) * 2 + x + 1 + 20 * x];

    float top_left_12 = input[block_start_idx + (thread_id / 4) * 2 + 22 * x];
    float top_right_12 = input[block_start_idx + (thread_id / 4) * 2 + 1 + 22 * x];
    float bottom_left_12 = input[block_start_idx + (thread_id / 4) * 2 + x + 22 * x];
    float bottom_right_12 = input[block_start_idx + (thread_id / 4) * 2 + x + 1 + 22 * x];

    float top_left_13 = input[block_start_idx + (thread_id / 4) * 2 + 24 * x];
    float top_right_13 = input[block_start_idx + (thread_id / 4) * 2 + 1 + 24 * x];
    float bottom_left_13 = input[block_start_idx + (thread_id / 4) * 2 + x + 24 * x];
    float bottom_right_13 = input[block_start_idx + (thread_id / 4) * 2 + x + 1 + 24 * x];

    float top_left_14 = input[block_start_idx + (thread_id / 4) * 2 + 26 * x];
    float top_right_14 = input[block_start_idx + (thread_id / 4) * 2 + 1 + 26 * x];
    float bottom_left_14 = input[block_start_idx + (thread_id / 4) * 2 + x + 26 * x];
    float bottom_right_14 = input[block_start_idx + (thread_id / 4) * 2 + x + 1 + 26 * x];

    float top_left_15 = input[block_start_idx + (thread_id / 4) * 2 + 28 * x];
    float top_right_15 = input[block_start_idx + (thread_id / 4) * 2 + 1 + 28 * x];
    float bottom_left_15 = input[block_start_idx + (thread_id / 4) * 2 + x + 28 * x];
    float bottom_right_15 = input[block_start_idx + (thread_id / 4) * 2 + x + 1 + 28 * x];

    float top_left_16 = input[block_start_idx + (thread_id / 4) * 2 + 30 * x];
    float top_right_16 = input[block_start_idx + (thread_id / 4) * 2 + 1 + 30 * x];
    float bottom_left_16 = input[block_start_idx + (thread_id / 4) * 2 + x + 30 * x];
    float bottom_right_16 = input[block_start_idx + (thread_id / 4) * 2 + x + 1 + 30 * x];

    //__device__ float fmaf ( float  x, float  y, float  z )
    float tempOutRow1 = 0.0f;
    float tempOutRow2 = 0.0f;
    float tempOutRow3 = 0.0f;
    float tempOutRow4 = 0.0f;
    float tempOutRow5 = 0.0f;
    float tempOutRow6 = 0.0f;
    float tempOutRow7 = 0.0f;
    float tempOutRow8 = 0.0f;
    float tempOutRow9 = 0.0f;
    float tempOutRow10 = 0.0f;
    float tempOutRow11 = 0.0f;
    float tempOutRow12 = 0.0f;
    float tempOutRow13 = 0.0f;
    float tempOutRow14 = 0.0f;
    float tempOutRow15 = 0.0f;
    float tempOutRow16 = 0.0f;

    float tempOutRow17 = 0.0f;
    float tempOutRow18 = 0.0f;
    float tempOutRow19 = 0.0f;
    float tempOutRow20 = 0.0f;
    float tempOutRow21 = 0.0f;
    float tempOutRow22 = 0.0f;
    float tempOutRow23 = 0.0f;
    float tempOutRow24 = 0.0f;
    float tempOutRow25 = 0.0f;
    float tempOutRow26 = 0.0f;
    float tempOutRow27 = 0.0f;
    float tempOutRow28 = 0.0f;
    float tempOutRow29 = 0.0f;
    float tempOutRow30 = 0.0f;
    float tempOutRow31 = 0.0f;
    float tempOutRow32 = 0.0f;

    float tempOutRow33 = 0.0f;
    float tempOutRow34 = 0.0f;
    float tempOutRow35 = 0.0f;
    float tempOutRow36 = 0.0f;
    float tempOutRow37 = 0.0f;
    float tempOutRow38 = 0.0f;
    float tempOutRow39 = 0.0f;
    float tempOutRow40 = 0.0f;
    float tempOutRow41 = 0.0f;
    float tempOutRow42 = 0.0f;
    float tempOutRow43 = 0.0f;
    float tempOutRow44 = 0.0f;
    float tempOutRow45 = 0.0f;
    float tempOutRow46 = 0.0f;
    float tempOutRow47 = 0.0f;
    float tempOutRow48 = 0.0f;
    
    float tempOutRow49 = 0.0f;
    float tempOutRow50 = 0.0f;
    float tempOutRow51 = 0.0f;
    float tempOutRow52 = 0.0f;
    float tempOutRow53 = 0.0f;
    float tempOutRow54 = 0.0f;
    float tempOutRow55 = 0.0f;
    float tempOutRow56 = 0.0f;
    float tempOutRow57 = 0.0f;
    float tempOutRow58 = 0.0f;
    float tempOutRow59 = 0.0f;
    float tempOutRow60 = 0.0f;
    float tempOutRow61 = 0.0f;
    float tempOutRow62 = 0.0f;
    float tempOutRow63 = 0.0f;
    float tempOutRow64 = 0.0f;

    // Top_Left Partial Sums for all rows
    tempOutRow1 = fmaf(top_left_1, gtop_left_coeff[index], tempOutRow1);
    tempOutRow2 = fmaf(top_left_1, gtop_left_coeff[index + 4], tempOutRow2);
    tempOutRow3 = fmaf(top_left_1, gtop_left_coeff[index + 8], tempOutRow3);
    tempOutRow4 = fmaf(top_left_1, gtop_left_coeff[index + 12], tempOutRow4);
    tempOutRow5 = fmaf(top_left_2, gtop_left_coeff[index], tempOutRow5);
    tempOutRow6 = fmaf(top_left_2, gtop_left_coeff[index + 4], tempOutRow6);
    tempOutRow7 = fmaf(top_left_2, gtop_left_coeff[index + 8], tempOutRow7);
    tempOutRow8 = fmaf(top_left_2, gtop_left_coeff[index + 12], tempOutRow8);
    tempOutRow9 = fmaf(top_left_3, gtop_left_coeff[index], tempOutRow9);
    tempOutRow10 = fmaf(top_left_3, gtop_left_coeff[index + 4], tempOutRow10);
    tempOutRow11 = fmaf(top_left_3, gtop_left_coeff[index + 8], tempOutRow11);
    tempOutRow12 = fmaf(top_left_3, gtop_left_coeff[index + 12], tempOutRow12);
    tempOutRow13 = fmaf(top_left_4, gtop_left_coeff[index], tempOutRow13);
    tempOutRow14 = fmaf(top_left_4, gtop_left_coeff[index + 4], tempOutRow14);
    tempOutRow15 = fmaf(top_left_4, gtop_left_coeff[index + 8], tempOutRow15);
    tempOutRow16 = fmaf(top_left_4, gtop_left_coeff[index + 12], tempOutRow16);
    
    tempOutRow17 = fmaf(top_left_5, gtop_left_coeff[index], tempOutRow17);
    tempOutRow18 = fmaf(top_left_5, gtop_left_coeff[index + 4], tempOutRow18);
    tempOutRow19 = fmaf(top_left_5, gtop_left_coeff[index + 8], tempOutRow19);
    tempOutRow20 = fmaf(top_left_5, gtop_left_coeff[index + 12], tempOutRow20);
    tempOutRow21 = fmaf(top_left_6, gtop_left_coeff[index], tempOutRow21);
    tempOutRow22 = fmaf(top_left_6, gtop_left_coeff[index + 4], tempOutRow22);
    tempOutRow23 = fmaf(top_left_6, gtop_left_coeff[index + 8], tempOutRow23);
    tempOutRow24 = fmaf(top_left_6, gtop_left_coeff[index + 12], tempOutRow24);
    tempOutRow25 = fmaf(top_left_7, gtop_left_coeff[index], tempOutRow25);
    tempOutRow26 = fmaf(top_left_7, gtop_left_coeff[index + 4], tempOutRow26);
    tempOutRow27 = fmaf(top_left_7, gtop_left_coeff[index + 8], tempOutRow27);
    tempOutRow28 = fmaf(top_left_7, gtop_left_coeff[index + 12], tempOutRow28);
    tempOutRow29 = fmaf(top_left_8, gtop_left_coeff[index], tempOutRow29);
    tempOutRow30 = fmaf(top_left_8, gtop_left_coeff[index + 4], tempOutRow30);
    tempOutRow31 = fmaf(top_left_8, gtop_left_coeff[index + 8], tempOutRow31);
    tempOutRow32 = fmaf(top_left_8, gtop_left_coeff[index + 12], tempOutRow32);

    tempOutRow33 = fmaf(top_left_9, gtop_left_coeff[index], tempOutRow33);
    tempOutRow34 = fmaf(top_left_9, gtop_left_coeff[index + 4], tempOutRow34);
    tempOutRow35 = fmaf(top_left_9, gtop_left_coeff[index + 8], tempOutRow35);
    tempOutRow36 = fmaf(top_left_9, gtop_left_coeff[index + 12], tempOutRow36);
    tempOutRow37 = fmaf(top_left_10, gtop_left_coeff[index], tempOutRow37);
    tempOutRow38 = fmaf(top_left_10, gtop_left_coeff[index + 4], tempOutRow38);
    tempOutRow39 = fmaf(top_left_10, gtop_left_coeff[index + 8], tempOutRow39);
    tempOutRow40 = fmaf(top_left_10, gtop_left_coeff[index + 12], tempOutRow40);
    tempOutRow41 = fmaf(top_left_11, gtop_left_coeff[index], tempOutRow41);
    tempOutRow42 = fmaf(top_left_11, gtop_left_coeff[index + 4], tempOutRow42);
    tempOutRow43 = fmaf(top_left_11, gtop_left_coeff[index + 8], tempOutRow43);
    tempOutRow44 = fmaf(top_left_11, gtop_left_coeff[index + 12], tempOutRow44);
    tempOutRow45 = fmaf(top_left_12, gtop_left_coeff[index], tempOutRow45);
    tempOutRow46 = fmaf(top_left_12, gtop_left_coeff[index + 4], tempOutRow46);
    tempOutRow47 = fmaf(top_left_12, gtop_left_coeff[index + 8], tempOutRow47);
    tempOutRow48 = fmaf(top_left_12, gtop_left_coeff[index + 12], tempOutRow48);
    
    tempOutRow49 = fmaf(top_left_13, gtop_left_coeff[index], tempOutRow49);
    tempOutRow50 = fmaf(top_left_13, gtop_left_coeff[index + 4], tempOutRow50);
    tempOutRow51 = fmaf(top_left_13, gtop_left_coeff[index + 8], tempOutRow51);
    tempOutRow52 = fmaf(top_left_13, gtop_left_coeff[index + 12], tempOutRow52);
    tempOutRow53 = fmaf(top_left_14, gtop_left_coeff[index], tempOutRow53);
    tempOutRow54 = fmaf(top_left_14, gtop_left_coeff[index + 4], tempOutRow54);
    tempOutRow55 = fmaf(top_left_14, gtop_left_coeff[index + 8], tempOutRow55);
    tempOutRow56 = fmaf(top_left_14, gtop_left_coeff[index + 12], tempOutRow56);
    tempOutRow57 = fmaf(top_left_15, gtop_left_coeff[index], tempOutRow57);
    tempOutRow58 = fmaf(top_left_15, gtop_left_coeff[index + 4], tempOutRow58);
    tempOutRow59 = fmaf(top_left_15, gtop_left_coeff[index + 8], tempOutRow59);
    tempOutRow60 = fmaf(top_left_15, gtop_left_coeff[index + 12], tempOutRow60);
    tempOutRow61 = fmaf(top_left_16, gtop_left_coeff[index], tempOutRow61);
    tempOutRow62 = fmaf(top_left_16, gtop_left_coeff[index + 4], tempOutRow62);
    tempOutRow63 = fmaf(top_left_16, gtop_left_coeff[index + 8], tempOutRow63);
    tempOutRow64 = fmaf(top_left_16, gtop_left_coeff[index + 12], tempOutRow64);

    // Top_Right Partial Sums for all rows
    tempOutRow1 = fmaf(top_right_1, gtop_right_coeff[index], tempOutRow1);
    tempOutRow2 = fmaf(top_right_1, gtop_right_coeff[index + 4], tempOutRow2);
    tempOutRow3 = fmaf(top_right_1, gtop_right_coeff[index + 8], tempOutRow3);
    tempOutRow4 = fmaf(top_right_1, gtop_right_coeff[index + 12], tempOutRow4);
    tempOutRow5 = fmaf(top_right_2, gtop_right_coeff[index], tempOutRow5);
    tempOutRow6 = fmaf(top_right_2, gtop_right_coeff[index + 4], tempOutRow6);
    tempOutRow7 = fmaf(top_right_2, gtop_right_coeff[index + 8], tempOutRow7);
    tempOutRow8 = fmaf(top_right_2, gtop_right_coeff[index + 12], tempOutRow8);
    tempOutRow9 = fmaf(top_right_3, gtop_right_coeff[index], tempOutRow9);
    tempOutRow10 = fmaf(top_right_3, gtop_right_coeff[index + 4], tempOutRow10);
    tempOutRow11 = fmaf(top_right_3, gtop_right_coeff[index + 8], tempOutRow11);
    tempOutRow12 = fmaf(top_right_3, gtop_right_coeff[index + 12], tempOutRow12);
    tempOutRow13 = fmaf(top_right_4, gtop_right_coeff[index], tempOutRow13);
    tempOutRow14 = fmaf(top_right_4, gtop_right_coeff[index + 4], tempOutRow14);
    tempOutRow15 = fmaf(top_right_4, gtop_right_coeff[index + 8], tempOutRow15);
    tempOutRow16 = fmaf(top_right_4, gtop_right_coeff[index + 12], tempOutRow16);
    
    tempOutRow17 = fmaf(top_right_5, gtop_right_coeff[index], tempOutRow17);
    tempOutRow18 = fmaf(top_right_5, gtop_right_coeff[index + 4], tempOutRow18);
    tempOutRow19 = fmaf(top_right_5, gtop_right_coeff[index + 8], tempOutRow19);
    tempOutRow20 = fmaf(top_right_5, gtop_right_coeff[index + 12], tempOutRow20);
    tempOutRow21 = fmaf(top_right_6, gtop_right_coeff[index], tempOutRow21);
    tempOutRow22 = fmaf(top_right_6, gtop_right_coeff[index + 4], tempOutRow22);
    tempOutRow23 = fmaf(top_right_6, gtop_right_coeff[index + 8], tempOutRow23);
    tempOutRow24 = fmaf(top_right_6, gtop_right_coeff[index + 12], tempOutRow24);
    tempOutRow25 = fmaf(top_right_7, gtop_right_coeff[index], tempOutRow25);
    tempOutRow26 = fmaf(top_right_7, gtop_right_coeff[index + 4], tempOutRow26);
    tempOutRow27 = fmaf(top_right_7, gtop_right_coeff[index + 8], tempOutRow27);
    tempOutRow28 = fmaf(top_right_7, gtop_right_coeff[index + 12], tempOutRow28);
    tempOutRow29 = fmaf(top_right_8, gtop_right_coeff[index], tempOutRow29);
    tempOutRow30 = fmaf(top_right_8, gtop_right_coeff[index + 4], tempOutRow30);
    tempOutRow31 = fmaf(top_right_8, gtop_right_coeff[index + 8], tempOutRow31);
    tempOutRow32 = fmaf(top_right_8, gtop_right_coeff[index + 12], tempOutRow32);

    tempOutRow33 = fmaf(top_right_9, gtop_right_coeff[index], tempOutRow33);
    tempOutRow34 = fmaf(top_right_9, gtop_right_coeff[index + 4], tempOutRow34);
    tempOutRow35 = fmaf(top_right_9, gtop_right_coeff[index + 8], tempOutRow35);
    tempOutRow36 = fmaf(top_right_9, gtop_right_coeff[index + 12], tempOutRow36);
    tempOutRow37 = fmaf(top_right_10, gtop_right_coeff[index], tempOutRow37);
    tempOutRow38 = fmaf(top_right_10, gtop_right_coeff[index + 4], tempOutRow38);
    tempOutRow39 = fmaf(top_right_10, gtop_right_coeff[index + 8], tempOutRow39);
    tempOutRow40 = fmaf(top_right_10, gtop_right_coeff[index + 12], tempOutRow40);
    tempOutRow41 = fmaf(top_right_11, gtop_right_coeff[index], tempOutRow41);
    tempOutRow42 = fmaf(top_right_11, gtop_right_coeff[index + 4], tempOutRow42);
    tempOutRow43 = fmaf(top_right_11, gtop_right_coeff[index + 8], tempOutRow43);
    tempOutRow44 = fmaf(top_right_11, gtop_right_coeff[index + 12], tempOutRow44);
    tempOutRow45 = fmaf(top_right_12, gtop_right_coeff[index], tempOutRow45);
    tempOutRow46 = fmaf(top_right_12, gtop_right_coeff[index + 4], tempOutRow46);
    tempOutRow47 = fmaf(top_right_12, gtop_right_coeff[index + 8], tempOutRow47);
    tempOutRow48 = fmaf(top_right_12, gtop_right_coeff[index + 12], tempOutRow48);
    
    tempOutRow49 = fmaf(top_right_13, gtop_right_coeff[index], tempOutRow49);
    tempOutRow50 = fmaf(top_right_13, gtop_right_coeff[index + 4], tempOutRow50);
    tempOutRow51 = fmaf(top_right_13, gtop_right_coeff[index + 8], tempOutRow51);
    tempOutRow52 = fmaf(top_right_13, gtop_right_coeff[index + 12], tempOutRow52);
    tempOutRow53 = fmaf(top_right_14, gtop_right_coeff[index], tempOutRow53);
    tempOutRow54 = fmaf(top_right_14, gtop_right_coeff[index + 4], tempOutRow54);
    tempOutRow55 = fmaf(top_right_14, gtop_right_coeff[index + 8], tempOutRow55);
    tempOutRow56 = fmaf(top_right_14, gtop_right_coeff[index + 12], tempOutRow56);
    tempOutRow57 = fmaf(top_right_15, gtop_right_coeff[index], tempOutRow57);
    tempOutRow58 = fmaf(top_right_15, gtop_right_coeff[index + 4], tempOutRow58);
    tempOutRow59 = fmaf(top_right_15, gtop_right_coeff[index + 8], tempOutRow59);
    tempOutRow60 = fmaf(top_right_15, gtop_right_coeff[index + 12], tempOutRow60);
    tempOutRow61 = fmaf(top_right_16, gtop_right_coeff[index], tempOutRow61);
    tempOutRow62 = fmaf(top_right_16, gtop_right_coeff[index + 4], tempOutRow62);
    tempOutRow63 = fmaf(top_right_16, gtop_right_coeff[index + 8], tempOutRow63);
    tempOutRow64 = fmaf(top_right_16, gtop_right_coeff[index + 12], tempOutRow64);

    // Bottom_Left Partial Sums for all rows
    tempOutRow1 = fmaf(bottom_left_1, gbottom_left_coeff[index], tempOutRow1);
    tempOutRow2 = fmaf(bottom_left_1, gbottom_left_coeff[index + 4], tempOutRow2);
    tempOutRow3 = fmaf(bottom_left_1, gbottom_left_coeff[index + 8], tempOutRow3);
    tempOutRow4 = fmaf(bottom_left_1, gbottom_left_coeff[index + 12], tempOutRow4);
    tempOutRow5 = fmaf(bottom_left_2, gbottom_left_coeff[index], tempOutRow5);
    tempOutRow6 = fmaf(bottom_left_2, gbottom_left_coeff[index + 4], tempOutRow6);
    tempOutRow7 = fmaf(bottom_left_2, gbottom_left_coeff[index + 8], tempOutRow7);
    tempOutRow8 = fmaf(bottom_left_2, gbottom_left_coeff[index + 12], tempOutRow8);
    tempOutRow9 = fmaf(bottom_left_3, gbottom_left_coeff[index], tempOutRow9);
    tempOutRow10 = fmaf(bottom_left_3, gbottom_left_coeff[index + 4], tempOutRow10);
    tempOutRow11 = fmaf(bottom_left_3, gbottom_left_coeff[index + 8], tempOutRow11);
    tempOutRow12 = fmaf(bottom_left_3, gbottom_left_coeff[index + 12], tempOutRow12);
    tempOutRow13 = fmaf(bottom_left_4, gbottom_left_coeff[index], tempOutRow13);
    tempOutRow14 = fmaf(bottom_left_4, gbottom_left_coeff[index + 4], tempOutRow14);
    tempOutRow15 = fmaf(bottom_left_4, gbottom_left_coeff[index + 8], tempOutRow15);
    tempOutRow16 = fmaf(bottom_left_4, gbottom_left_coeff[index + 12], tempOutRow16);

    tempOutRow17 = fmaf(bottom_left_5, gbottom_left_coeff[index], tempOutRow17);
    tempOutRow18 = fmaf(bottom_left_5, gbottom_left_coeff[index + 4], tempOutRow18);
    tempOutRow19 = fmaf(bottom_left_5, gbottom_left_coeff[index + 8], tempOutRow19);
    tempOutRow20 = fmaf(bottom_left_5, gbottom_left_coeff[index + 12], tempOutRow20);
    tempOutRow21 = fmaf(bottom_left_6, gbottom_left_coeff[index], tempOutRow21);
    tempOutRow22 = fmaf(bottom_left_6, gbottom_left_coeff[index + 4], tempOutRow22);
    tempOutRow23 = fmaf(bottom_left_6, gbottom_left_coeff[index + 8], tempOutRow23);
    tempOutRow24 = fmaf(bottom_left_6, gbottom_left_coeff[index + 12], tempOutRow24);
    tempOutRow25 = fmaf(bottom_left_7, gbottom_left_coeff[index], tempOutRow25);
    tempOutRow26 = fmaf(bottom_left_7, gbottom_left_coeff[index + 4], tempOutRow26);
    tempOutRow27 = fmaf(bottom_left_7, gbottom_left_coeff[index + 8], tempOutRow27);
    tempOutRow28 = fmaf(bottom_left_7, gbottom_left_coeff[index + 12], tempOutRow28);
    tempOutRow29 = fmaf(bottom_left_8, gbottom_left_coeff[index], tempOutRow29);
    tempOutRow30 = fmaf(bottom_left_8, gbottom_left_coeff[index + 4], tempOutRow30);
    tempOutRow31 = fmaf(bottom_left_8, gbottom_left_coeff[index + 8], tempOutRow31);
    tempOutRow32 = fmaf(bottom_left_8, gbottom_left_coeff[index + 12], tempOutRow32);

    tempOutRow33 = fmaf(bottom_left_9, gbottom_left_coeff[index], tempOutRow33);
    tempOutRow34 = fmaf(bottom_left_9, gbottom_left_coeff[index + 4], tempOutRow34);
    tempOutRow35 = fmaf(bottom_left_9, gbottom_left_coeff[index + 8], tempOutRow35);
    tempOutRow36 = fmaf(bottom_left_9, gbottom_left_coeff[index + 12], tempOutRow36);
    tempOutRow37 = fmaf(bottom_left_10, gbottom_left_coeff[index], tempOutRow37);
    tempOutRow38 = fmaf(bottom_left_10, gbottom_left_coeff[index + 4], tempOutRow38);
    tempOutRow39 = fmaf(bottom_left_10, gbottom_left_coeff[index + 8], tempOutRow39);
    tempOutRow40 = fmaf(bottom_left_10, gbottom_left_coeff[index + 12], tempOutRow40);
    tempOutRow41 = fmaf(bottom_left_11, gbottom_left_coeff[index], tempOutRow41);
    tempOutRow42 = fmaf(bottom_left_11, gbottom_left_coeff[index + 4], tempOutRow42);
    tempOutRow43 = fmaf(bottom_left_11, gbottom_left_coeff[index + 8], tempOutRow43);
    tempOutRow44 = fmaf(bottom_left_11, gbottom_left_coeff[index + 12], tempOutRow44);
    tempOutRow45 = fmaf(bottom_left_12, gbottom_left_coeff[index], tempOutRow45);
    tempOutRow46 = fmaf(bottom_left_12, gbottom_left_coeff[index + 4], tempOutRow46);
    tempOutRow47 = fmaf(bottom_left_12, gbottom_left_coeff[index + 8], tempOutRow47);
    tempOutRow48 = fmaf(bottom_left_12, gbottom_left_coeff[index + 12], tempOutRow48);
    
    tempOutRow49 = fmaf(bottom_left_13, gbottom_left_coeff[index], tempOutRow49);
    tempOutRow50 = fmaf(bottom_left_13, gbottom_left_coeff[index + 4], tempOutRow50);
    tempOutRow51 = fmaf(bottom_left_13, gbottom_left_coeff[index + 8], tempOutRow51);
    tempOutRow52 = fmaf(bottom_left_13, gbottom_left_coeff[index + 12], tempOutRow52);
    tempOutRow53 = fmaf(bottom_left_14, gbottom_left_coeff[index], tempOutRow53);
    tempOutRow54 = fmaf(bottom_left_14, gbottom_left_coeff[index + 4], tempOutRow54);
    tempOutRow55 = fmaf(bottom_left_14, gbottom_left_coeff[index + 8], tempOutRow55);
    tempOutRow56 = fmaf(bottom_left_14, gbottom_left_coeff[index + 12], tempOutRow56);
    tempOutRow57 = fmaf(bottom_left_15, gbottom_left_coeff[index], tempOutRow57);
    tempOutRow58 = fmaf(bottom_left_15, gbottom_left_coeff[index + 4], tempOutRow58);
    tempOutRow59 = fmaf(bottom_left_15, gbottom_left_coeff[index + 8], tempOutRow59);
    tempOutRow60 = fmaf(bottom_left_15, gbottom_left_coeff[index + 12], tempOutRow60);
    tempOutRow61 = fmaf(bottom_left_16, gbottom_left_coeff[index], tempOutRow61);
    tempOutRow62 = fmaf(bottom_left_16, gbottom_left_coeff[index + 4], tempOutRow62);
    tempOutRow63 = fmaf(bottom_left_16, gbottom_left_coeff[index + 8], tempOutRow63);
    tempOutRow64 = fmaf(bottom_left_16, gbottom_left_coeff[index + 12], tempOutRow64);

    // Bottom_Right Partial Sums for all rows
    tempOutRow1 = fmaf(bottom_right_1, gbottom_right_coeff[index], tempOutRow1);
    tempOutRow2 = fmaf(bottom_right_1, gbottom_right_coeff[index + 4], tempOutRow2);
    tempOutRow3 = fmaf(bottom_right_1, gbottom_right_coeff[index + 8], tempOutRow3);
    tempOutRow4 = fmaf(bottom_right_1, gbottom_right_coeff[index + 12], tempOutRow4);
    tempOutRow5 = fmaf(bottom_right_2, gbottom_right_coeff[index], tempOutRow5);
    tempOutRow6 = fmaf(bottom_right_2, gbottom_right_coeff[index + 4], tempOutRow6);
    tempOutRow7 = fmaf(bottom_right_2, gbottom_right_coeff[index + 8], tempOutRow7);
    tempOutRow8 = fmaf(bottom_right_2, gbottom_right_coeff[index + 12], tempOutRow8);
    tempOutRow9 = fmaf(bottom_right_3, gbottom_right_coeff[index], tempOutRow9);
    tempOutRow10 = fmaf(bottom_right_3, gbottom_right_coeff[index + 4], tempOutRow10);
    tempOutRow11 = fmaf(bottom_right_3, gbottom_right_coeff[index + 8], tempOutRow11);
    tempOutRow12 = fmaf(bottom_right_3, gbottom_right_coeff[index + 12], tempOutRow12);
    tempOutRow13 = fmaf(bottom_right_4, gbottom_right_coeff[index], tempOutRow13);
    tempOutRow14 = fmaf(bottom_right_4, gbottom_right_coeff[index + 4], tempOutRow14);
    tempOutRow15 = fmaf(bottom_right_4, gbottom_right_coeff[index + 8], tempOutRow15);
    tempOutRow16 = fmaf(bottom_right_4, gbottom_right_coeff[index + 12], tempOutRow16);

    tempOutRow17 = fmaf(bottom_right_5, gbottom_right_coeff[index], tempOutRow17);
    tempOutRow18 = fmaf(bottom_right_5, gbottom_right_coeff[index + 4], tempOutRow18);
    tempOutRow19 = fmaf(bottom_right_5, gbottom_right_coeff[index + 8], tempOutRow19);
    tempOutRow20 = fmaf(bottom_right_5, gbottom_right_coeff[index + 12], tempOutRow20);
    tempOutRow21 = fmaf(bottom_right_6, gbottom_right_coeff[index], tempOutRow21);
    tempOutRow22 = fmaf(bottom_right_6, gbottom_right_coeff[index + 4], tempOutRow22);
    tempOutRow23 = fmaf(bottom_right_6, gbottom_right_coeff[index + 8], tempOutRow23);
    tempOutRow24 = fmaf(bottom_right_6, gbottom_right_coeff[index + 12], tempOutRow24);
    tempOutRow25 = fmaf(bottom_right_7, gbottom_right_coeff[index], tempOutRow25);
    tempOutRow26 = fmaf(bottom_right_7, gbottom_right_coeff[index + 4], tempOutRow26);
    tempOutRow27 = fmaf(bottom_right_7, gbottom_right_coeff[index + 8], tempOutRow27);
    tempOutRow28 = fmaf(bottom_right_7, gbottom_right_coeff[index + 12], tempOutRow28);
    tempOutRow29 = fmaf(bottom_right_8, gbottom_right_coeff[index], tempOutRow29);
    tempOutRow30 = fmaf(bottom_right_8, gbottom_right_coeff[index + 4], tempOutRow30);
    tempOutRow31 = fmaf(bottom_right_8, gbottom_right_coeff[index + 8], tempOutRow31);
    tempOutRow32 = fmaf(bottom_right_8, gbottom_right_coeff[index + 12], tempOutRow32);

    tempOutRow33 = fmaf(bottom_right_9, gbottom_right_coeff[index], tempOutRow33);
    tempOutRow34 = fmaf(bottom_right_9, gbottom_right_coeff[index + 4], tempOutRow34);
    tempOutRow35 = fmaf(bottom_right_9, gbottom_right_coeff[index + 8], tempOutRow35);
    tempOutRow36 = fmaf(bottom_right_9, gbottom_right_coeff[index + 12], tempOutRow36);
    tempOutRow37 = fmaf(bottom_right_10, gbottom_right_coeff[index], tempOutRow37);
    tempOutRow38 = fmaf(bottom_right_10, gbottom_right_coeff[index + 4], tempOutRow38);
    tempOutRow39 = fmaf(bottom_right_10, gbottom_right_coeff[index + 8], tempOutRow39);
    tempOutRow40 = fmaf(bottom_right_10, gbottom_right_coeff[index + 12], tempOutRow40);
    tempOutRow41 = fmaf(bottom_right_11, gbottom_right_coeff[index], tempOutRow41);
    tempOutRow42 = fmaf(bottom_right_11, gbottom_right_coeff[index + 4], tempOutRow42);
    tempOutRow43 = fmaf(bottom_right_11, gbottom_right_coeff[index + 8], tempOutRow43);
    tempOutRow44 = fmaf(bottom_right_11, gbottom_right_coeff[index + 12], tempOutRow44);
    tempOutRow45 = fmaf(bottom_right_12, gbottom_right_coeff[index], tempOutRow45);
    tempOutRow46 = fmaf(bottom_right_12, gbottom_right_coeff[index + 4], tempOutRow46);
    tempOutRow47 = fmaf(bottom_right_12, gbottom_right_coeff[index + 8], tempOutRow47);
    tempOutRow48 = fmaf(bottom_right_12, gbottom_right_coeff[index + 12], tempOutRow48);
    
    tempOutRow49 = fmaf(bottom_right_13, gbottom_right_coeff[index], tempOutRow49);
    tempOutRow50 = fmaf(bottom_right_13, gbottom_right_coeff[index + 4], tempOutRow50);
    tempOutRow51 = fmaf(bottom_right_13, gbottom_right_coeff[index + 8], tempOutRow51);
    tempOutRow52 = fmaf(bottom_right_13, gbottom_right_coeff[index + 12], tempOutRow52);
    tempOutRow53 = fmaf(bottom_right_14, gbottom_right_coeff[index], tempOutRow53);
    tempOutRow54 = fmaf(bottom_right_14, gbottom_right_coeff[index + 4], tempOutRow54);
    tempOutRow55 = fmaf(bottom_right_14, gbottom_right_coeff[index + 8], tempOutRow55);
    tempOutRow56 = fmaf(bottom_right_14, gbottom_right_coeff[index + 12], tempOutRow56);
    tempOutRow57 = fmaf(bottom_right_15, gbottom_right_coeff[index], tempOutRow57);
    tempOutRow58 = fmaf(bottom_right_15, gbottom_right_coeff[index + 4], tempOutRow58);
    tempOutRow59 = fmaf(bottom_right_15, gbottom_right_coeff[index + 8], tempOutRow59);
    tempOutRow60 = fmaf(bottom_right_15, gbottom_right_coeff[index + 12], tempOutRow60);
    tempOutRow61 = fmaf(bottom_right_16, gbottom_right_coeff[index], tempOutRow61);
    tempOutRow62 = fmaf(bottom_right_16, gbottom_right_coeff[index + 4], tempOutRow62);
    tempOutRow63 = fmaf(bottom_right_16, gbottom_right_coeff[index + 8], tempOutRow63);
    tempOutRow64 = fmaf(bottom_right_16, gbottom_right_coeff[index + 12], tempOutRow64);

    output[block_start_idx_out + thread_id] = tempOutRow1;
    output[block_start_idx_out + thread_id + 2*x] = tempOutRow2;
    output[block_start_idx_out + thread_id + 4*x] = tempOutRow3;
    output[block_start_idx_out + thread_id + 6*x] = tempOutRow4;
    output[block_start_idx_out + thread_id + 8*x] = tempOutRow5;
    output[block_start_idx_out + thread_id + 10*x] = tempOutRow6;
    output[block_start_idx_out + thread_id + 12*x] = tempOutRow7;
    output[block_start_idx_out + thread_id + 14*x] = tempOutRow8;
    output[block_start_idx_out + thread_id + 16*x] = tempOutRow9;
    output[block_start_idx_out + thread_id + 18*x] = tempOutRow10;
    output[block_start_idx_out + thread_id + 20*x] = tempOutRow11;
    output[block_start_idx_out + thread_id + 22*x] = tempOutRow12;
    output[block_start_idx_out + thread_id + 24*x] = tempOutRow13;
    output[block_start_idx_out + thread_id + 26*x] = tempOutRow14;
    output[block_start_idx_out + thread_id + 28*x] = tempOutRow15;
    output[block_start_idx_out + thread_id + 30*x] = tempOutRow16;
    
    output[block_start_idx_out + thread_id + 32*x] = tempOutRow17;
    output[block_start_idx_out + thread_id + 34*x] = tempOutRow18;
    output[block_start_idx_out + thread_id + 36*x] = tempOutRow19;
    output[block_start_idx_out + thread_id + 38*x] = tempOutRow20;
    output[block_start_idx_out + thread_id + 40*x] = tempOutRow21;
    output[block_start_idx_out + thread_id + 42*x] = tempOutRow22;
    output[block_start_idx_out + thread_id + 44*x] = tempOutRow23;
    output[block_start_idx_out + thread_id + 46*x] = tempOutRow24;
    output[block_start_idx_out + thread_id + 48*x] = tempOutRow25;
    output[block_start_idx_out + thread_id + 50*x] = tempOutRow26;
    output[block_start_idx_out + thread_id + 52*x] = tempOutRow27;
    output[block_start_idx_out + thread_id + 54*x] = tempOutRow28;
    output[block_start_idx_out + thread_id + 56*x] = tempOutRow29;
    output[block_start_idx_out + thread_id + 58*x] = tempOutRow30;
    output[block_start_idx_out + thread_id + 60*x] = tempOutRow31;
    output[block_start_idx_out + thread_id + 62*x] = tempOutRow32;

    output[block_start_idx_out + thread_id + 64*x] = tempOutRow33;
    output[block_start_idx_out + thread_id + 66*x] = tempOutRow34;
    output[block_start_idx_out + thread_id + 68*x] = tempOutRow35;
    output[block_start_idx_out + thread_id + 70*x] = tempOutRow36;
    output[block_start_idx_out + thread_id + 72*x] = tempOutRow37;
    output[block_start_idx_out + thread_id + 74*x] = tempOutRow38;
    output[block_start_idx_out + thread_id + 76*x] = tempOutRow39;
    output[block_start_idx_out + thread_id + 78*x] = tempOutRow40;
    output[block_start_idx_out + thread_id + 80*x] = tempOutRow41;
    output[block_start_idx_out + thread_id + 82*x] = tempOutRow42;
    output[block_start_idx_out + thread_id + 84*x] = tempOutRow43;
    output[block_start_idx_out + thread_id + 86*x] = tempOutRow44;
    output[block_start_idx_out + thread_id + 88*x] = tempOutRow45;
    output[block_start_idx_out + thread_id + 90*x] = tempOutRow46;
    output[block_start_idx_out + thread_id + 92*x] = tempOutRow47;
    output[block_start_idx_out + thread_id + 94*x] = tempOutRow48;
    
    output[block_start_idx_out + thread_id + 96*x] = tempOutRow49;
    output[block_start_idx_out + thread_id + 98*x] = tempOutRow50;
    output[block_start_idx_out + thread_id + 100*x] = tempOutRow51;
    output[block_start_idx_out + thread_id + 102*x] = tempOutRow52;
    output[block_start_idx_out + thread_id + 104*x] = tempOutRow53;
    output[block_start_idx_out + thread_id + 106*x] = tempOutRow54;
    output[block_start_idx_out + thread_id + 108*x] = tempOutRow55;
    output[block_start_idx_out + thread_id + 110*x] = tempOutRow56;
    output[block_start_idx_out + thread_id + 112*x] = tempOutRow57;
    output[block_start_idx_out + thread_id + 114*x] = tempOutRow58;
    output[block_start_idx_out + thread_id + 116*x] = tempOutRow59;
    output[block_start_idx_out + thread_id + 118*x] = tempOutRow60;
    output[block_start_idx_out + thread_id + 120*x] = tempOutRow61;
    output[block_start_idx_out + thread_id + 122*x] = tempOutRow62;
    output[block_start_idx_out + thread_id + 124*x] = tempOutRow63;
    output[block_start_idx_out + thread_id + 126*x] = tempOutRow64;
}

void decodeImage(float *inputImageR, float *inputImageG, float *inputImageB, char *fileName){
    int index = 0;
    float *tmpBuffer;
    // READ IMAGE and Init buffers
    // const char *fileName = "inputs/2048x2048.jpg";
    Mat fullImage, windowImage;
    Mat channels[3];
    std::vector<float> array;
    fullImage = imread(fileName);
    int imageRows = (int)fullImage.rows, imageCols = (int)fullImage.cols;
    
    split(fullImage, channels);
    array.assign(channels[0].datastart, channels[0].dataend);
    tmpBuffer = &array[0];
    memcpy(inputImageB, tmpBuffer, imageCols * imageRows * sizeof(float));
    array.assign(channels[1].datastart, channels[1].dataend);
    tmpBuffer = &array[0];
    memcpy(inputImageG, tmpBuffer, imageCols * imageRows * sizeof(float));
    array.assign(channels[2].datastart, channels[2].dataend);
    tmpBuffer = &array[0];
    memcpy(inputImageR, tmpBuffer, imageCols * imageRows * sizeof(float));
}

void encodeImage(float *outputR, float *outputG, float *outputB){
    const char *fileName = "/afs/ece.cmu.edu/usr/arexhari/Public/645-project/results/128x128-bl-gpu.jpg";
    vector<Mat> channels;
    Mat finalImage;
    cv::Mat matR = cv::Mat(INPUTHEIGHT*2, INPUTWIDTH*2, CV_32F, outputR);
    cv::Mat matG = cv::Mat(INPUTHEIGHT*2, INPUTWIDTH*2, CV_32F, outputG);
    cv::Mat matB = cv::Mat(INPUTHEIGHT*2, INPUTWIDTH*2, CV_32F, outputB);

    channels.push_back(matB);
    channels.push_back(matG);
    channels.push_back(matR);

    merge(channels, finalImage);
    imwrite(fileName, finalImage); 
}

int main(int argc, char **argv){
    // read in input 2x2 pixels
    char *fileName = argv[1];
    INPUTHEIGHT = atoi(argv[3]);
    INPUTWIDTH = atoi(argv[2]);
    float *inputImageR = (float *)calloc(INPUTWIDTH * INPUTHEIGHT, sizeof(float));
    float *inputImageG = (float *)calloc(INPUTWIDTH * INPUTHEIGHT, sizeof(float)); 
    float *inputImageB = (float *)calloc(INPUTWIDTH * INPUTHEIGHT, sizeof(float));

    decodeImage(inputImageR, inputImageG, inputImageB, fileName);

    float *dev_in_red, *dev_out_red, *dev_in_green, *dev_out_green, *dev_in_blue, *dev_out_blue;
    float *host_out_red, *host_out_green, *host_out_blue;

    host_out_red = (float*) malloc(INPUTHEIGHT * INPUTWIDTH * 4 * sizeof(float));
    host_out_green = (float*) malloc(INPUTHEIGHT * INPUTWIDTH * 4 * sizeof(float));
    host_out_blue = (float*) malloc(INPUTHEIGHT * INPUTWIDTH * 4 * sizeof(float));

    hipError_t err = hipMalloc(&dev_in_red, INPUTHEIGHT * INPUTWIDTH *sizeof(float));
    if (err != hipSuccess){
      cout<<"Dev Memory not allocated"<<endl;
      exit(-1);
    }

    err = hipMalloc(&dev_in_green, INPUTHEIGHT * INPUTWIDTH *sizeof(float));
    if (err != hipSuccess){
      cout<<"Dev Memory not allocated"<<endl;
      exit(-1);
    }

    err = hipMalloc(&dev_in_blue, INPUTHEIGHT * INPUTWIDTH *sizeof(float));
    if (err != hipSuccess){
      cout<<"Dev Memory not allocated"<<endl;
      exit(-1);
    }

    err = hipMalloc(&dev_out_red, INPUTHEIGHT * INPUTWIDTH * 4 *sizeof(float));
    if (err != hipSuccess){
      cout<<"Dev Memory not allocated"<<endl;
      exit(-1);
    }

    err = hipMalloc(&dev_out_green, INPUTHEIGHT * INPUTWIDTH * 4 *sizeof(float));
    if (err != hipSuccess){
      cout<<"Dev Memory not allocated"<<endl;
      exit(-1);
    }

    err = hipMalloc(&dev_out_blue, INPUTHEIGHT * INPUTWIDTH * 4 *sizeof(float));
    if (err != hipSuccess){
      cout<<"Dev Memory not allocated"<<endl;
      exit(-1);
    }
    
    hipMemcpy(dev_in_red, inputImageR, INPUTHEIGHT * INPUTWIDTH * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_in_green, inputImageG, INPUTHEIGHT * INPUTWIDTH * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_in_blue, inputImageB, INPUTHEIGHT * INPUTWIDTH * sizeof(float), hipMemcpyHostToDevice);

    //create GPU timing events for timing the GPU
    hipEvent_t st2, et2;
    hipEventCreate(&st2);
    hipEventCreate(&et2);        
  float milliseconds = 9999, millisecondsI = 0;
    int block_num = (INPUTHEIGHT * INPUTWIDTH) / (32 * 128);
  for(int l = 0; l < 10; l++){
    hipEventRecord(st2);
    base_kernel<<<block_num, 256>>>(dev_in_red, dev_out_red, INPUTWIDTH, INPUTHEIGHT);
    base_kernel<<<block_num, 256>>>(dev_in_green, dev_out_green, INPUTWIDTH, INPUTHEIGHT);
    base_kernel<<<block_num, 256>>>(dev_in_blue, dev_out_blue, INPUTWIDTH, INPUTHEIGHT);
    hipEventRecord(et2);
        
    //host waits until et2 has occured     
    hipEventSynchronize(et2);
    hipEventElapsedTime(&millisecondsI, st2, et2);
    if (millisecondsI < milliseconds){
      milliseconds = millisecondsI;
    }
  }

    cout<<INPUTHEIGHT << "x" << INPUTWIDTH << " "<<milliseconds<<"ms"<<endl;

    hipMemcpy(host_out_red, dev_out_red, INPUTHEIGHT * INPUTWIDTH * 4 * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(host_out_green, dev_out_green, INPUTHEIGHT * INPUTWIDTH * 4 * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(host_out_blue, dev_out_blue, INPUTHEIGHT * INPUTWIDTH * 4 * sizeof(float), hipMemcpyDeviceToHost);

    // encodeImage(host_out_red, host_out_green, host_out_blue);

    return 0;
}