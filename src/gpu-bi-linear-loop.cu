#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <sstream>
#include <stdio.h>
#include <stdlib.h>
#include <cstdlib>
#include "opencv2/opencv.hpp"

using namespace cv;
using namespace std;

// __constant__ float gtop_left_coeff[16];
// __constant__ float gtop_right_coeff[16];
// __constant__ float gbottom_left_coeff[16];
// __constant__ float gbottom_right_coeff[16];

static int INPUTHEIGHT;
static int INPUTWIDTH;

__global__ void base_kernel(float *input, float *output, int x, int y) {
    int thread_id = threadIdx.x; //get thread id; 0-127
    int block_id = blockIdx.x; // get block id;

    // gtop_left_coeff = {1, 2 / 3.0, 1 / 3.0, 0, 2 / 3.0, 4 / 9.0, 2 / 9.0, 0, 1 / 3.0, 2 / 9.0, 1 / 9.0, 0, 0, 0, 0, 0};
    float gtop_left_coeff[16];
    gtop_left_coeff[0] = 1;
    gtop_left_coeff[1] = 2 / 3.0;
    gtop_left_coeff[2] = 1 / 3.0;
    gtop_left_coeff[3] = 0;
    gtop_left_coeff[4] = 2 / 3.0;
    gtop_left_coeff[5] = 4 / 9.0;
    gtop_left_coeff[6] = 2 / 9.0;
    gtop_left_coeff[7] = 0;
    gtop_left_coeff[8] = 1 / 3.0;
    gtop_left_coeff[9] = 2 / 9.0;
    gtop_left_coeff[10] = 1 / 9.0;
    gtop_left_coeff[11] = 0;
    gtop_left_coeff[12] = 0;
    gtop_left_coeff[13] = 0;
    gtop_left_coeff[14] = 0;
    gtop_left_coeff[15] = 0;
    

    // gtop_right_coeff[16] = {0, 1 / 3.0, 2 / 3.0, 1, 0, 2 / 9.0, 4 / 9.0, 2 / 3.0, 0, 1 / 9.0, 2 / 9.0, 1 / 3.0, 0, 0, 0, 0};
    float gtop_right_coeff[16];
    gtop_right_coeff[0] = 0;
    gtop_right_coeff[1] = 1 / 3.0;
    gtop_right_coeff[2] = 2 / 3.0;
    gtop_right_coeff[3] = 1;
    gtop_right_coeff[4] = 0;
    gtop_right_coeff[5] = 2 / 9.0;
    gtop_right_coeff[6] = 4 / 9.0;
    gtop_right_coeff[7] = 2 / 3.0;
    gtop_right_coeff[8] = 0;
    gtop_right_coeff[9] = 1 / 9.0;
    gtop_right_coeff[10] = 2 / 9.0;
    gtop_right_coeff[11] = 1 / 3.0;
    gtop_right_coeff[12] = 0;
    gtop_right_coeff[13] = 0;
    gtop_right_coeff[14] = 0;
    gtop_right_coeff[15] = 0;
    
    // float gbottom_left_coeff[] = {0, 0, 0, 0, 1 / 3.0, 2 / 9.0, 1 / 9.0, 0, 2 / 3.0, 4 / 9.0, 2 / 9.0, 0, 1, 2 / 3.0, 1 / 3.0, 0};
    float gbottom_left_coeff[16];
    gbottom_left_coeff[0] = 0;
    gbottom_left_coeff[1] = 0;
    gbottom_left_coeff[2] = 0;
    gbottom_left_coeff[3] = 0;
    gbottom_left_coeff[4] = 1/3.0;
    gbottom_left_coeff[5] = 2/9.0;
    gbottom_left_coeff[6] = 1/9.0;
    gbottom_left_coeff[7] = 0;
    gbottom_left_coeff[8] = 2 / 3.0;
    gbottom_left_coeff[9] = 4 / 9.0;
    gbottom_left_coeff[10] = 2 / 9.0;
    gbottom_left_coeff[11] = 0;
    gbottom_left_coeff[12] = 1;
    gbottom_left_coeff[13] = 2 / 3.0;
    gbottom_left_coeff[14] = 1 / 3.0;
    gbottom_left_coeff[15] = 0;

    // float gbottom_right_coeff[] = {0, 0, 0, 0, 0, 1 / 9.0, 2 / 9.0, 1 / 3.0, 0, 2 / 9.0, 4 / 9.0, 2 / 3.0, 0, 1 / 3.0, 2 / 3.0, 1};
    float gbottom_right_coeff[16];
    gbottom_right_coeff[0] = 0;
    gbottom_right_coeff[1] = 0;
    gbottom_right_coeff[2] = 0;
    gbottom_right_coeff[3] = 0;
    gbottom_right_coeff[4] = 0;
    gbottom_right_coeff[5] = 1 / 9.0;
    gbottom_right_coeff[6] = 2 / 9.0;
    gbottom_right_coeff[7] = 1 / 3.0;
    gbottom_right_coeff[8] = 2 / 9.0;
    gbottom_right_coeff[9] = 2 / 9.0;
    gbottom_right_coeff[10] = 4 / 9.0;
    gbottom_right_coeff[11] = 2 / 3.0;
    gbottom_right_coeff[12] = 0;
    gbottom_right_coeff[13] = 1 / 3.0;
    gbottom_right_coeff[14] = 2 / 3.0;
    gbottom_right_coeff[15] = 1;
    
    // 2 x 64 -> 4 x 128
    // int kernel_number = thread_id / 4;
    // 4 input pixel: kernel_number * 2, kernel_number * 2 + 1, kernel_number * 2 + x, kernel_number * 2 + x + 1

    int num_blocks = blockDim.x;
    int num_blocks_per_row = x / 128;

    int block_row = block_id / num_blocks_per_row;
    int block_col = block_id % num_blocks_per_row;
    
    int block_start_idx = (block_row * 32) * (x) + block_col * 128; // input: 4 rows of length x per block
    int block_start_idx_out = (block_row * 64) * (x * 2) + block_col * 256; // output: 8 rows of length 2x per block

    // Base Kernel
    int index = thread_id % 4;
    for (int i = 0; i < 16; ++i) {
        float top_left = input[block_start_idx + (thread_id / 4) * 2 + (i*2*x)];
        float top_right = input[block_start_idx + (thread_id / 4) * 2 + 1 + (i*2*x)];
        float bottom_left = input[block_start_idx + (thread_id / 4) * 2 + x + (i*2*x)];
        float bottom_right = input[block_start_idx + (thread_id / 4) * 2 + x + 1 + (i*2*x)];

        //__device__ float fmaf ( float  x, float  y, float  z )
        float tempOutRow1 = 0.0f;
        float tempOutRow2 = 0.0f;
        float tempOutRow3 = 0.0f;
        float tempOutRow4 = 0.0f;

        // Top_Left Partial Sums for all rows
        tempOutRow1 = fmaf(top_left, gtop_left_coeff[index], tempOutRow1);
        tempOutRow2 = fmaf(top_left, gtop_left_coeff[index + 4], tempOutRow2);
        tempOutRow3 = fmaf(top_left, gtop_left_coeff[index + 8], tempOutRow3);
        tempOutRow4 = fmaf(top_left, gtop_left_coeff[index + 12], tempOutRow4);

        // Top_Right Partial Sums for all rows
        tempOutRow1 = fmaf(top_right, gtop_right_coeff[index], tempOutRow1);
        tempOutRow2 = fmaf(top_right, gtop_right_coeff[index + 4], tempOutRow2);
        tempOutRow3 = fmaf(top_right, gtop_right_coeff[index + 8], tempOutRow3);
        tempOutRow4 = fmaf(top_right, gtop_right_coeff[index + 12], tempOutRow4);

        // Bottom_Left Partial Sums for all rows
        tempOutRow1 = fmaf(bottom_left, gbottom_left_coeff[index], tempOutRow1);
        tempOutRow2 = fmaf(bottom_left, gbottom_left_coeff[index + 4], tempOutRow2);
        tempOutRow3 = fmaf(bottom_left, gbottom_left_coeff[index + 8], tempOutRow3);
        tempOutRow4 = fmaf(bottom_left, gbottom_left_coeff[index + 12], tempOutRow4);

        // Bottom_Right Partial Sums for all rows
        tempOutRow1 = fmaf(bottom_right, gbottom_right_coeff[index], tempOutRow1);
        tempOutRow2 = fmaf(bottom_right, gbottom_right_coeff[index + 4], tempOutRow2);
        tempOutRow3 = fmaf(bottom_right, gbottom_right_coeff[index + 8], tempOutRow3);
        tempOutRow4 = fmaf(bottom_right, gbottom_right_coeff[index + 12], tempOutRow4);

        output[block_start_idx_out + thread_id + (i * 4 * 2 * x)] = tempOutRow1;
        output[block_start_idx_out + thread_id + 2*x + (i * 4 * 2 * x)] = tempOutRow2;
        output[block_start_idx_out + thread_id + 4*x + (i * 4 * 2 * x)] = tempOutRow3;
        output[block_start_idx_out + thread_id + 6*x + (i * 4 * 2 * x)] = tempOutRow4;
    }
}

void decodeImage(float *inputImageR, float *inputImageG, float *inputImageB, char *fileName){
    int index = 0;
    float *tmpBuffer;
    // READ IMAGE and Init buffers
    // const char *fileName = "inputs/2048x2048.jpg";
    Mat fullImage, windowImage;
    Mat channels[3];
    std::vector<float> array;
    fullImage = imread(fileName);
    int imageRows = (int)fullImage.rows, imageCols = (int)fullImage.cols;
    
    split(fullImage, channels);
    array.assign(channels[0].datastart, channels[0].dataend);
    tmpBuffer = &array[0];
    memcpy(inputImageB, tmpBuffer, imageCols * imageRows * sizeof(float));
    array.assign(channels[1].datastart, channels[1].dataend);
    tmpBuffer = &array[0];
    memcpy(inputImageG, tmpBuffer, imageCols * imageRows * sizeof(float));
    array.assign(channels[2].datastart, channels[2].dataend);
    tmpBuffer = &array[0];
    memcpy(inputImageR, tmpBuffer, imageCols * imageRows * sizeof(float));
}

void encodeImage(float *outputR, float *outputG, float *outputB){
    const char *fileName = "/afs/ece.cmu.edu/usr/arexhari/Public/645-project/results/128x128-bl-gpu.jpg";
    vector<Mat> channels;
    Mat finalImage;
    cv::Mat matR = cv::Mat(INPUTHEIGHT*2, INPUTWIDTH*2, CV_32F, outputR);
    cv::Mat matG = cv::Mat(INPUTHEIGHT*2, INPUTWIDTH*2, CV_32F, outputG);
    cv::Mat matB = cv::Mat(INPUTHEIGHT*2, INPUTWIDTH*2, CV_32F, outputB);

    channels.push_back(matB);
    channels.push_back(matG);
    channels.push_back(matR);

    merge(channels, finalImage);
    imwrite(fileName, finalImage); 
}


int main(int argc, char **argv){
    // read in input 2x2 pixels
    char *fileName = argv[1];
    INPUTHEIGHT = atoi(argv[3]);
    INPUTWIDTH = atoi(argv[2]);
    float *inputImageR = (float *)calloc(INPUTWIDTH * INPUTHEIGHT, sizeof(float));
    float *inputImageG = (float *)calloc(INPUTWIDTH * INPUTHEIGHT, sizeof(float)); 
    float *inputImageB = (float *)calloc(INPUTWIDTH * INPUTHEIGHT, sizeof(float));

    decodeImage(inputImageR, inputImageG, inputImageB, fileName);

    float *dev_in_red, *dev_out_red, *dev_in_green, *dev_out_green, *dev_in_blue, *dev_out_blue;
    float *host_out_red, *host_out_green, *host_out_blue;

    host_out_red = (float*) malloc(INPUTHEIGHT * INPUTWIDTH * 4 * sizeof(float));
    host_out_green = (float*) malloc(INPUTHEIGHT * INPUTWIDTH * 4 * sizeof(float));
    host_out_blue = (float*) malloc(INPUTHEIGHT * INPUTWIDTH * 4 * sizeof(float));

    hipError_t err = hipMalloc(&dev_in_red, INPUTHEIGHT * INPUTWIDTH *sizeof(float));
    if (err != hipSuccess){
      cout<<"Dev Memory not allocated"<<endl;
      exit(-1);
    }

    err = hipMalloc(&dev_in_green, INPUTHEIGHT * INPUTWIDTH *sizeof(float));
    if (err != hipSuccess){
      cout<<"Dev Memory not allocated"<<endl;
      exit(-1);
    }

    err = hipMalloc(&dev_in_blue, INPUTHEIGHT * INPUTWIDTH *sizeof(float));
    if (err != hipSuccess){
      cout<<"Dev Memory not allocated"<<endl;
      exit(-1);
    }

    err = hipMalloc(&dev_out_red, INPUTHEIGHT * INPUTWIDTH * 4 *sizeof(float));
    if (err != hipSuccess){
      cout<<"Dev Memory not allocated"<<endl;
      exit(-1);
    }

    err = hipMalloc(&dev_out_green, INPUTHEIGHT * INPUTWIDTH * 4 *sizeof(float));
    if (err != hipSuccess){
      cout<<"Dev Memory not allocated"<<endl;
      exit(-1);
    }

    err = hipMalloc(&dev_out_blue, INPUTHEIGHT * INPUTWIDTH * 4 *sizeof(float));
    if (err != hipSuccess){
      cout<<"Dev Memory not allocated"<<endl;
      exit(-1);
    }
    
    hipMemcpy(dev_in_red, inputImageR, INPUTHEIGHT * INPUTWIDTH * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_in_green, inputImageG, INPUTHEIGHT * INPUTWIDTH * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_in_blue, inputImageB, INPUTHEIGHT * INPUTWIDTH * sizeof(float), hipMemcpyHostToDevice);

    //create GPU timing events for timing the GPU
    hipEvent_t st2, et2;
    hipEventCreate(&st2);
    hipEventCreate(&et2);        
    float milliseconds = 9999, millisecondsI = 0;
    int block_num = (INPUTHEIGHT * INPUTWIDTH) / (32 * 128);
  for(int l = 0; l < 10; l++){
    hipEventRecord(st2);
    base_kernel<<<block_num, 256>>>(dev_in_red, dev_out_red, INPUTWIDTH, INPUTHEIGHT);
    base_kernel<<<block_num, 256>>>(dev_in_green, dev_out_green, INPUTWIDTH, INPUTHEIGHT);
    base_kernel<<<block_num, 256>>>(dev_in_blue, dev_out_blue, INPUTWIDTH, INPUTHEIGHT);
    hipEventRecord(et2);
        
    //host waits until et2 has occured     
    hipEventSynchronize(et2);
    hipEventElapsedTime(&millisecondsI, st2, et2);
    if (millisecondsI < milliseconds){
      milliseconds = millisecondsI;
    }
  }

    cout<<INPUTHEIGHT << "x" << INPUTWIDTH << " "<<milliseconds<<"ms"<<endl;

    hipMemcpy(host_out_red, dev_out_red, INPUTHEIGHT * INPUTWIDTH * 4 * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(host_out_green, dev_out_green, INPUTHEIGHT * INPUTWIDTH * 4 * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(host_out_blue, dev_out_blue, INPUTHEIGHT * INPUTWIDTH * 4 * sizeof(float), hipMemcpyDeviceToHost);

    // encodeImage(host_out_red, host_out_green, host_out_blue);

    return 0;
}